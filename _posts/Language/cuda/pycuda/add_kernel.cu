
#include <hip/hip_runtime.h>
// add_kernel.cu

__global__ void add_kernel(float *a, float *b, float *c, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}