#include "hip/hip_runtime.h"
#include "matmul.cuh"

#include "hip/hip_runtime.h"
#include ""

#include <chrono>
#include <cmath>

#include <stdio.h>
#include <stdlib.h>

__global__ void matMul(int* dA, int* dB, int* dC, int m, int n, int k)
{
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	int col = blockDim.y * blockIdx.y + threadIdx.y;
	int index = row * n + col;
	if (row >= m || col >= n)
	{
		return;
	}
	int sum = 0;
	for (int i = 0; i < k; ++i)
	{
		sum += dA[row * k + i] * dB[n * i + col];
	}
	dC[index] = sum;
}

void matMulWrapper(int* A, int* B, int* C, int m, int n, int k)
{
    std::chrono::system_clock::time_point start = std::chrono::system_clock::now();
    int* dA, * dB, * dC;
	hipMalloc(&dA, m * k * sizeof(int));
	hipMemset(dA, 0, m * k * sizeof(int));
	hipMalloc(&dB, k * n * sizeof(int));
	hipMemset(dB, 0, k * n * sizeof(int));
	hipMalloc(&dC, m * n * sizeof(int));
	hipMemset(dC, 0, m * n * sizeof(int));
    std::chrono::duration<double> timeCudaMalloc = std::chrono::system_clock::now() - start;

    start = std::chrono::system_clock::now();
	hipMemcpy(dA, A, m * k * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dB, B, k * n * sizeof(int), hipMemcpyHostToDevice);
	std::chrono::duration<double> timeCudaMemcpyToDevice = std::chrono::system_clock::now() - start;

	int blockSize = 32;
	dim3 gridDim(ceil(static_cast<float>(m) / blockSize), ceil(static_cast<float>(n) / blockSize));
	dim3 blockDim(blockSize, blockSize);
	printf("Grid(%d, %d), Block(%d, %d)\n", gridDim.x, gridDim.y, blockDim.x, blockDim.y);

    start = std::chrono::system_clock::now();
	matMul <<<gridDim, blockDim >>> (dA, dB, dC, m, n, k);
	hipDeviceSynchronize();
    std::chrono::duration<double> timeCudaMatmul = std::chrono::system_clock::now() - start;

    start = std::chrono::system_clock::now();
	hipMemcpy(C, dC, m * n * sizeof(int), hipMemcpyDeviceToHost);
    std::chrono::duration<double> timeCudaMemcpyToHost = std::chrono::system_clock::now() - start;

    hipFree(dA);
	hipFree(dB);
	hipFree(dC);

    printf("cuda malloc : %lf(ms), cuda memcpy to device : %lf(ms), cuda matmul : %lf(ms), cuda memcpy to host : %lf(ms)\n", timeCudaMalloc*1000, timeCudaMemcpyToDevice*1000, timeCudaMatmul*1000, timeCudaMemcpyToHost*1000);   
}