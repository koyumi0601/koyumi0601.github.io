#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for (int deviceIdx = 0; deviceIdx < deviceCount; ++deviceIdx) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, deviceIdx);

        std::cout << "Device " << deviceIdx << ": " << deviceProp.name << std::endl;
        std::cout << "  Compute Capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "  Multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << "  CUDA Cores per Multiprocessor: " << deviceProp.warpSize << std::endl;
        std::cout << "  Global Memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "  Shared Memory per Block: " << deviceProp.sharedMemPerBlock / 1024 << " KB" << std::endl;
        std::cout << "  Max Threads per Block: " << deviceProp.maxThreadsPerBlock << std::endl;
        std::cout << "  Max Threads per Dimension: (" << deviceProp.maxThreadsDim[0] << ", "
                  << deviceProp.maxThreadsDim[1] << ", " << deviceProp.maxThreadsDim[2] << ")" << std::endl;
        std::cout << "  Max Grid Size: (" << deviceProp.maxGridSize[0] << ", " << deviceProp.maxGridSize[1] << ", "
                  << deviceProp.maxGridSize[2] << ")" << std::endl;
        // Tensor Core 지원 여부 확인
        if (deviceProp.major >= 7 && deviceProp.minor >= 0) {
            std::cout << "  Tensor Core 지원: Yes" << std::endl;
        } else {
            std::cout << "  Tensor Core 지원: No" << std::endl;
        }          

        std::cout << std::endl;
    }

    return 0;
}