#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

// Kernel function to add two vectors on GPU
__global__ void addKernel(int* c, const int* a, const int* b, int size)
{
  int i = threadIdx.x;
  if (i < size)
    c[i] = a[i] + b[i];
}

// Helper function for using CUDA to add vectors in parallel.
void addWithCuda(int* c, const int* a, const int* b, int size)
{
  int* dev_a = 0;
  int* dev_b = 0;
  int* dev_c = 0;

  // Allocate GPU memory for a, b, and c
  hipMalloc((void**)&dev_a, size * sizeof(int));
  hipMalloc((void**)&dev_b, size * sizeof(int));
  hipMalloc((void**)&dev_c, size * sizeof(int));

  // Copy input vectors from host memory to GPU buffers
  hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

  // Launch a kernel on the GPU with one thread for each element.
  addKernel << <1, size >> > (dev_c, dev_a, dev_b, size);

  // Copy result from GPU buffer to host memory
  hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

  // Free GPU memory
  hipFree(dev_c);
  hipFree(dev_a);
  hipFree(dev_b);
}

int main()
{
  const int arraySize = 5;
  const int a[arraySize] = { 1, 2, 3, 4, 5 };
  const int b[arraySize] = { 10, 20, 30, 40, 50 };
  int c[arraySize] = { 0 };

  addWithCuda(c, a, b, arraySize);

  printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
    c[0], c[1], c[2], c[3], c[4]);

  return 0;
}