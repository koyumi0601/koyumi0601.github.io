#include "hip/hip_runtime.h"
// kernel.cu
#include "hip/hip_runtime.h"
#include ""

__global__ void addKernel(int* c, const int* a, const int* b, unsigned int size)
{
  int i = threadIdx.x;
  if (i < size) {
    c[i] = a[i] + b[i];
  }
}

extern "C" __declspec(dllexport) void addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
  int* dev_a = 0;
  int* dev_b = 0;
  int* dev_c = 0;

  hipMalloc((void**)&dev_a, size * sizeof(int));
  hipMalloc((void**)&dev_b, size * sizeof(int));
  hipMalloc((void**)&dev_c, size * sizeof(int));

  hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

  addKernel << <1, size >> > (dev_c, dev_a, dev_b, size);

  hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
}