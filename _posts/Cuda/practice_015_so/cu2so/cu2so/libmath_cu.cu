#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>
// #define DLLEXPORT extern "C" __declspec(dllexport) // window


__global__ void mul_const_kernel(float* pddst, float* pdsrc, float dconst, int* pnsz) {
  int nx = pnsz[0];
  int ny = pnsz[1];
  int nz = pnsz[2];
  int id = 0;

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int idy = blockDim.y * blockIdx.y + threadIdx.y;
  int idz = blockDim.z * blockIdx.z + threadIdx.z;

  if (idz >= nz || idy >= ny || idx >= nx) return;

  id = ny * nx * idz + nx * idy + idx;
  pddst[id] = pdsrc[id] * dconst;


  return;
}


__global__ void add_const_kernel(float* pddst, float* pdsrc, float dconst, int* pnsz) {
  int nx = pnsz[0];
  int ny = pnsz[1];
  int nz = pnsz[2];
  int id = 0;

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int idy = blockDim.y * blockIdx.y + threadIdx.y;
  int idz = blockDim.z * blockIdx.z + threadIdx.z;

  if (idz >= nz || idy >= ny || idx >= nx) return;

  id = ny * nx * idz + nx * idy + idx;
  pddst[id] = pdsrc[id] + dconst;

  return;

}


__global__ void mul_mat_kernel(float* pddst, float* pdsrc, float* pdsrc2, int* pnsz) {
  int nx = pnsz[0];
  int ny = pnsz[1];
  int nz = pnsz[2];
  int id = 0;

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int idy = blockDim.y * blockIdx.y + threadIdx.y;
  int idz = blockDim.z * blockIdx.z + threadIdx.z;

  if (idz >= nz || idy >= ny || idx >= nx) return;

  id = ny * nx * idz + nx * idy + idx;
  pddst[id] = pdsrc[id] * pdsrc2[id];

  return;

}


__global__ void add_mat_kernel(float* pddst, float* pdsrc, float* pdsrc2, int* pnsz) {
  int nx = pnsz[0];
  int ny = pnsz[1];
  int nz = pnsz[2];
  int id = 0;

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int idy = blockDim.y * blockIdx.y + threadIdx.y;
  int idz = blockDim.z * blockIdx.z + threadIdx.z;

  if (idz >= nz || idy >= ny || idx >= nx) return;

  id = ny * nx * idz + nx * idy + idx;
  pddst[id] = pdsrc[id] + pdsrc2[id];

  return;

}

// cpu interface
// DLLEXPORT void mul_const(float *pddst, float *pdsrc, float dconst, int *pnsz){ // window only expression
extern "C" void mul_const(float* pddst, float* pdsrc, float dconst, int* pnsz) { // available on linux, window
  float* gpddst = 0;
  float* gpdsrc = 0;
  int* gpnsz = 0;

  hipMalloc((void**)&gpddst, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float));
  hipMalloc((void**)&gpdsrc, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float));
  hipMalloc((void**)&gpnsz, 3 * sizeof(int));

  hipMemset(gpddst, 0, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float));
  hipMemset(gpdsrc, 0, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float));
  hipMemset(gpnsz, 0, 3 * sizeof(float));

  hipMemcpy(gpdsrc, pdsrc, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float), hipMemcpyHostToDevice); // destination, source, memory size, direction
  hipMemcpy(gpnsz, pnsz, 3 * sizeof(int), hipMemcpyHostToDevice); // destination, source, memory size, direction

  int nthread = 8;
  dim3 nblock(nthread, nthread, nthread);
  dim3 ngrid((pnsz[0] + nthread - 1) / nthread,
    (pnsz[1] + nthread - 1) / nthread,
    (pnsz[2] + nthread - 1) / nthread);

  //mul_const_kernel <<<ngrid, nblock >>> (gpddst, gpdsrc, dconst, gpnsz);
  mul_const_kernel << <ngrid, nblock >> > (gpddst, gpdsrc, dconst, gpnsz);

  hipMemcpy(pddst, gpddst, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(gpddst);
  hipFree(gpdsrc);
  hipFree(gpnsz);

  gpddst = 0;
  gpdsrc = 0;
  gpnsz = 0;

  return;
}

// DLLEXPORT void add_const(float *pddst, float *pdsrc, float dconst, int *pnsz){ // window only
extern "C" void add_const(float* pddst, float* pdsrc, float dconst, int* pnsz) { // available on window and linux 
  float* gpddst = 0;
  float* gpdsrc = 0;
  int* gpnsz = 0;

  hipMalloc((void**)&gpddst, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float));
  hipMalloc((void**)&gpdsrc, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float));
  hipMalloc((void**)&gpnsz, 3 * sizeof(int));

  hipMemset(gpddst, 0, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float));
  hipMemset(gpdsrc, 0, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float));
  hipMemset(gpnsz, 0, 3 * sizeof(float));

  hipMemcpy(gpdsrc, pdsrc, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float), hipMemcpyHostToDevice); // destination, source, memory size, direction
  hipMemcpy(gpnsz, pnsz, 3 * sizeof(int), hipMemcpyHostToDevice); // destination, source, memory size, direction

  int nthread = 8;
  dim3 nblock(nthread, nthread, nthread);
  dim3 ngrid((pnsz[0] + nthread - 1) / nthread,
    (pnsz[1] + nthread - 1) / nthread,
    (pnsz[2] + nthread - 1) / nthread);

  add_const_kernel << <ngrid, nblock >> > (gpddst, gpdsrc, dconst, gpnsz);

  hipMemcpy(pddst, gpddst, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(gpddst);
  hipFree(gpdsrc);
  hipFree(gpnsz);

  gpddst = 0;
  gpdsrc = 0;
  gpnsz = 0;

  return;
}

// DLLEXPORT void mul_mat(float *pddst, float *pdsrc1, float *pdsrc2, int *pnsz){ // window only
extern "C" void mul_mat(float* pddst, float* pdsrc1, float* pdsrc2, int* pnsz) { // available on window and linux
  float* gpddst = 0;
  float* gpdsrc1 = 0;
  float* gpdsrc2 = 0;
  int* gpnsz = 0;

  hipMalloc((void**)&gpddst, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float));
  hipMalloc((void**)&gpdsrc1, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float));
  hipMalloc((void**)&gpdsrc2, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float));
  hipMalloc((void**)&gpnsz, 3 * sizeof(int));


  hipMemset(gpddst, 0, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float));
  hipMemset(gpdsrc1, 0, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float));
  hipMemset(gpdsrc2, 0, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float));
  hipMemset(gpnsz, 0, 3 * sizeof(int));

  hipMemcpy(gpdsrc1, pdsrc1, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float), hipMemcpyHostToDevice); // destination, source, memory size, direction
  hipMemcpy(gpdsrc2, pdsrc2, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float), hipMemcpyHostToDevice); // destination, source, memory size, direction
  hipMemcpy(gpnsz, pnsz, 3 * sizeof(int), hipMemcpyHostToDevice); // destination, source, memory size, direction

  int nthread = 8;
  dim3 nblock(nthread, nthread, nthread);
  dim3 ngrid((pnsz[0] + nthread - 1) / nthread,
    (pnsz[1] + nthread - 1) / nthread,
    (pnsz[2] + nthread - 1) / nthread);

  mul_mat_kernel << <ngrid, nblock >> > (gpddst, gpdsrc1, gpdsrc2, gpnsz);

  hipMemcpy(pddst, gpddst, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(gpddst);
  hipFree(gpdsrc1);
  hipFree(gpdsrc2);
  hipFree(gpnsz);

  gpddst = 0;
  gpdsrc1 = 0;
  gpdsrc2 = 0;
  gpnsz = 0;

  return;
}


// DLLEXPORT void add_mat(float *pddst, float *pdsrc1, float *pdsrc2, int *pnsz){ // window only
extern "C" void add_mat(float* pddst, float* pdsrc1, float* pdsrc2, int* pnsz) { // available on window and linux
  float* gpddst = 0;
  float* gpdsrc1 = 0;
  float* gpdsrc2 = 0;
  int* gpnsz = 0;

  hipMalloc((void**)&gpddst, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float));
  hipMalloc((void**)&gpdsrc1, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float));
  hipMalloc((void**)&gpdsrc2, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float));
  hipMalloc((void**)&gpnsz, 3 * sizeof(int));


  hipMemset(gpddst, 0, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float));
  hipMemset(gpdsrc1, 0, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float));
  hipMemset(gpdsrc2, 0, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float));
  hipMemset(gpnsz, 0, 3 * sizeof(int));

  hipMemcpy(gpdsrc1, pdsrc1, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float), hipMemcpyHostToDevice); // destination, source, memory size, direction
  hipMemcpy(gpdsrc2, pdsrc2, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float), hipMemcpyHostToDevice); // destination, source, memory size, direction
  hipMemcpy(gpnsz, pnsz, 3 * sizeof(int), hipMemcpyHostToDevice); // destination, source, memory size, direction

  int nthread = 8;
  dim3 nblock(nthread, nthread, nthread);
  dim3 ngrid((pnsz[0] + nthread - 1) / nthread,
    (pnsz[1] + nthread - 1) / nthread,
    (pnsz[2] + nthread - 1) / nthread);

  add_mat_kernel << <ngrid, nblock >> > (gpddst, gpdsrc1, gpdsrc2, gpnsz);

  hipMemcpy(pddst, gpddst, pnsz[0] * pnsz[1] * pnsz[2] * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(gpddst);
  hipFree(gpdsrc1);
  hipFree(gpdsrc2);
  hipFree(gpnsz);

  gpddst = 0;
  gpdsrc1 = 0;
  gpdsrc2 = 0;
  gpnsz = 0;

  return;
}




// nvcc -Xcompiler -fPIC math_cu.cu -shared -o libmath_cu.so // linux
// nvcc -o math_cu_win.dll math_cu_linux.cu --shared // window
// nvcc error   : 'cudafe++' died with status 0xC0000005 (ACCESS_VIOLATION)



//
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
//
//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}
//
//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}
