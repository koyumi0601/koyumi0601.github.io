#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "kernel.cuh"
#include <numeric>
#include <iostream>


__global__ void averageVectorForKernels(unsigned char* deviceOutputPlane, unsigned char** deviceVecVolSlices, int numElements, int numVectors)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < numElements) {
    unsigned int sum = 0;
    for (int vec = 0; vec < numVectors; ++vec) {
      sum += deviceVecVolSlices[vec][i];
    }
    deviceOutputPlane[i] = sum / numVectors;
  }
}


hipError_t averageVectorForWithCuda(std::vector<unsigned char>& outputPlane, std::vector<unsigned char>& vecVol, unsigned int dim1Size, unsigned int dim2Size, unsigned int dim3Size) {
  unsigned char* deviceOutputPlane;
  std::vector<unsigned char*> deviceVecVolSlicesHost(dim3Size, nullptr); // Host-side vector slice pointers
  unsigned char** deviceVecVolSlices; // Device-side pointer array
  hipError_t cudaStatus;

  // Select GPU
  cudaStatus = hipSetDevice(0);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
    goto Error;
  }

  // Allocate GPU buffer
  cudaStatus = hipMalloc(&deviceOutputPlane, outputPlane.size() * sizeof(unsigned char));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed for deviceOutputPlane!");
    goto Error;
  }

  // Allocate memory for device-side pointer array
  cudaStatus = hipMalloc(&deviceVecVolSlices, sizeof(unsigned char*) * dim3Size);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed for deviceVecVolSlices!");
    goto Error;
  }

  for (int i = 0; i < dim3Size; ++i) {
    std::vector<unsigned char> vecVolSlice(vecVol.begin() + i * dim1Size * dim2Size, vecVol.begin() + (i + 1) * dim1Size * dim2Size);
    cudaStatus = hipMalloc(&deviceVecVolSlicesHost[i], vecVolSlice.size() * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipMalloc failed for deviceVecVolSlicesHost[%d]!", i);
      goto Error;
    }
    cudaStatus = hipMemcpy(deviceVecVolSlicesHost[i], vecVolSlice.data(), vecVolSlice.size() * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipMemcpy failed at host to device for slice %d!", i);
      goto Error;
    }
  }

  // Copy pointer array from host to device
  cudaStatus = hipMemcpy(deviceVecVolSlices, deviceVecVolSlicesHost.data(), sizeof(unsigned char*) * dim3Size, hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed for deviceVecVolSlices to device!");
    goto Error;
  }

  // Launch kernel
  dim3 threadsPerBlock(256);
  dim3 numBlocks((outputPlane.size() + threadsPerBlock.x - 1) / threadsPerBlock.x);
  averageVectorForKernels << <numBlocks, threadsPerBlock >> > (deviceOutputPlane, deviceVecVolSlices, dim1Size * dim2Size, dim3Size);

  // Check for kernel launch errors
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "averageVectorNKernels launch failed: %s\n", hipGetErrorString(cudaStatus));
    goto Error;
  }

  // Synchronize kernel and check for errors
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching averageVectorNKernels!\n", cudaStatus);
    goto Error;
  }

  // Copy output vector from GPU buffer to host memory
  outputPlane.resize(dim1Size * dim2Size);
  cudaStatus = hipMemcpy(outputPlane.data(), deviceOutputPlane, dim1Size * dim2Size * sizeof(unsigned char), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed for deviceOutputPlane to host!");
    goto Error;
  }

Error:
  hipFree(deviceOutputPlane);
  for (int i = 0; i < dim3Size; ++i) {
    hipFree(deviceVecVolSlicesHost[i]);
  }
  hipFree(deviceVecVolSlices);
  return cudaStatus;
}