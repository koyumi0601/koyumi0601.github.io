#include "hip/hip_runtime.h"
#include "kernel_bilinearinterpolation.cuh"

__global__ void generateIndexWiseRangeAngleMeshVecKernel
(
  float* deviceIndexWiseRangeMeshVecPtr,
  float* deviceIndexWiseAngleMeshVecPtr,
  double* deviceXPtr,
  double* deviceYPtr,
  int cols,
  int rows,
  float firstSrcRangeValue,
  float lastSrcRangeValue,
  int numRanges,
  float firstSrcAngleValue,
  float lastSrcAngleValue,
  int numAngles
)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < cols * rows)
  {
    int row = tid % rows;
    int col = tid / rows;
    int index = row + rows * col;
    float rMm = std::sqrt(std::pow((float)deviceXPtr[col], 2) + std::pow((float)deviceYPtr[row], 2));
    float thetaDeg = 90.0 - std::atan2((float)deviceYPtr[row], (float)deviceXPtr[col]) * 180.0 / PI; // atan2: angle to x axis.
    float IndexWiseRangeValue = (rMm - firstSrcRangeValue) * ((float)numRanges - 1.0f) / (lastSrcRangeValue - firstSrcRangeValue);
    if (IndexWiseRangeValue < 0 || IndexWiseRangeValue > ((float)numRanges - 1.0f))
    {
      deviceIndexWiseRangeMeshVecPtr[index] = -1.0;
    }
    else
    {
      deviceIndexWiseRangeMeshVecPtr[index] = IndexWiseRangeValue;
    }
    float IndexWiseAngleValue = (thetaDeg - firstSrcAngleValue) * ((float)numAngles - 1.0f) / (lastSrcAngleValue - firstSrcAngleValue);
    if (IndexWiseAngleValue < 0 || IndexWiseAngleValue > ((float)numAngles - 1.0f))
    {
      deviceIndexWiseAngleMeshVecPtr[index] = -1.0;
    }
    else
    {
      deviceIndexWiseAngleMeshVecPtr[index] = IndexWiseAngleValue;
    }
  }
}

__global__ void inverseScanConversionKernels
( unsigned char** deviceOutputVecVol,
  unsigned char** deviceVecVol,
  int srcRows,
  int srcCols,
  int numVectors,
  int dstRows,
  int dstCols,
  double* deviceIndexX,
  double* deviceIndexY
)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= dstRows * dstCols)
    return;

  int row = (int)deviceIndexY[idx];
  int col = (int)deviceIndexX[idx];
  if (row < 0 || col < 0 || row >= (srcRows - 1) || col >= (srcCols - 1))
  {
    for (int vec = 0; vec < numVectors; ++vec)
    {
      deviceOutputVecVol[vec][idx] = 0;
    }
  }
  else
  {
    double row_frac = deviceIndexY[idx] - (double)row;
    double col_frac = deviceIndexX[idx] - (double)col;
    for (int vec = 0; vec < numVectors; ++vec)
    {
      double intensity_topleft = (double)deviceVecVol[vec][row * srcCols + col];
      double intensity_topright = (double)deviceVecVol[vec][row * srcCols + col + 1];
      double intensity_bottomleft = (double)deviceVecVol[vec][(row + 1) * srcCols + col];
      double intensity_bottomright = (double)deviceVecVol[vec][(row + 1) * srcCols + col + 1];
      // Bilinear interpolation
      deviceOutputVecVol[vec][idx] = (int)(((1.0f - row_frac) * ((1.0f - col_frac) * intensity_topleft + col_frac * intensity_topright)
        + row_frac * ((1.0f - col_frac) * intensity_bottomleft + col_frac * intensity_bottomright)) + 0.5f);
    }
  }
}

std::pair<std::vector<float>, std::vector<float>> generateIndexWiseRangeAngleMeshVecWithCuda
( std::vector<double> X,
  std::vector<double> Y,
  std::vector<double> srcRangeA,
  std::vector<double> srcAngleA
)
{
  int cols = X.size();
  int rows = Y.size();
  int numSamplesPerFrame = cols * rows;
  int numRanges = srcRangeA.size();
  int numAngles = srcAngleA.size();
  float firstSrcRangeValue = (float) srcRangeA[0];
  float lastSrcRangeValue = (float) srcRangeA[srcRangeA.size() - 1];
  float firstSrcAngleValue = (float) srcAngleA[0];
  float lastSrcAngleValue = (float) srcAngleA[srcAngleA.size() - 1];
  std::vector<float> hostIndexWiseRangeMeshVec(X.size() * Y.size());
  std::vector<float> hostIndexWiseAngleMeshVec(X.size() * Y.size());
  double* deviceXPtr, * deviceYPtr;// , * deviceXMeshVecPtr, * deviceYMeshVecPtr, * deviceRangeMeshVecPtr, * deviceAngleMeshVecPtr;
  float* deviceIndexWiseRangeMeshVecPtr, * deviceIndexWiseAngleMeshVecPtr;
  hipMalloc(&deviceXPtr, cols * sizeof(double));
  hipMalloc(&deviceYPtr, rows * sizeof(double));
  hipMalloc(&deviceIndexWiseRangeMeshVecPtr, numSamplesPerFrame * sizeof(float));
  hipMalloc(&deviceIndexWiseAngleMeshVecPtr, numSamplesPerFrame * sizeof(float));
  hipMemcpy(deviceXPtr, X.data(), cols * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(deviceYPtr, Y.data(), rows * sizeof(double), hipMemcpyHostToDevice);
  int numThreads = 256; // 256 from 32 - 1024
  auto start = std::chrono::high_resolution_clock::now();
  int threadPerBlock = numThreads;
  dim3 blockDim(threadPerBlock, 1, 1);
  dim3 blockPerGrid = (numSamplesPerFrame + threadPerBlock - 1) / threadPerBlock;
  generateIndexWiseRangeAngleMeshVecKernel << <blockPerGrid, blockDim >> > (deviceIndexWiseRangeMeshVecPtr, deviceIndexWiseAngleMeshVecPtr, deviceXPtr, deviceYPtr, cols, rows, firstSrcRangeValue, lastSrcRangeValue, numRanges, firstSrcAngleValue, lastSrcAngleValue, numAngles);
  hipDeviceSynchronize();
  hipMemcpy(hostIndexWiseRangeMeshVec.data(), deviceIndexWiseRangeMeshVecPtr, numSamplesPerFrame * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(hostIndexWiseAngleMeshVec.data(), deviceIndexWiseAngleMeshVecPtr, numSamplesPerFrame * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(deviceXPtr);
  hipFree(deviceYPtr);
  hipFree(deviceIndexWiseRangeMeshVecPtr);
  hipFree(deviceIndexWiseAngleMeshVecPtr);
  return std::make_pair(hostIndexWiseRangeMeshVec, hostIndexWiseAngleMeshVec);
}

hipError_t inverseScanConversionWithCuda
( std::vector<unsigned char>& outputVecVol,
  std::vector<unsigned char>& nrrdData,
  unsigned int srcRows,
  unsigned int srcColumns,
  unsigned int numberOfFrames,
  unsigned int dstRows,
  unsigned int dstColumns,
  std::vector<double> srcIndexWiseXVec,
  std::vector<double> srcIndexWiseYVec
)
{
  hipError_t cudaStatus;
  double* deviceIndexX;
  double* deviceIndexY;
  std::vector<unsigned char*> deviceVecEachSlice(numberOfFrames, nullptr);
  unsigned char** deviceVecVol;
  unsigned char** deviceOutVol;
  unsigned char** hostVecEachSlicePtr = new unsigned char* [numberOfFrames];

  // Select GPU
  cudaStatus = hipSetDevice(0);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
    goto Error;
  }

  // malloc for deviceOutVol
  cudaStatus = hipMalloc(&deviceOutVol, sizeof(unsigned char*) * numberOfFrames);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed for deviceOutVol!");
    goto Error;
  }
  for (int i = 0; i < numberOfFrames; ++i)
  {
    std::vector<unsigned char> zeroVecSlice((size_t)dstRows * (size_t)dstColumns, 0);
    cudaStatus = hipMalloc(&deviceVecEachSlice[i], zeroVecSlice.size() * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipMalloc failed for deviceVecEachSlice[%d]!", i);
      goto Error;
    }
    cudaStatus = hipMemcpy(deviceVecEachSlice[i], zeroVecSlice.data(), zeroVecSlice.size() * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipMemcpy failed at host to device for slice %d!", i);
      goto Error;
    }
  }
  cudaStatus = hipMemcpy(deviceOutVol, deviceVecEachSlice.data(), sizeof(unsigned char*) * numberOfFrames, hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed for deviceOutVol to device!");
    goto Error;
  }


  // malloc and memcory for srcIndexWiseXVec and srcIndexWiseYVec
  cudaStatus = hipMalloc(&deviceIndexX, srcIndexWiseXVec.size() * sizeof(double));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed for deviceIndexX!");
    goto Error;
  }
  cudaStatus = hipMalloc(&deviceIndexY, srcIndexWiseYVec.size() * sizeof(double));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed for deviceIndexX!");
    goto Error;
  }
  cudaStatus = hipMemcpy(deviceIndexX, srcIndexWiseXVec.data(), srcIndexWiseXVec.size() * sizeof(double), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed for srcIndexWiseXVec to device!");
    goto Error;
  }
  cudaStatus = hipMemcpy(deviceIndexY, srcIndexWiseYVec.data(), srcIndexWiseYVec.size() * sizeof(double), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed for srcIndexWiseYVec to device!");
    goto Error;
  }

  // malloc and memcory for nrrddata
  cudaStatus = hipMalloc(&deviceVecVol, sizeof(unsigned char*) * numberOfFrames);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed for deviceVecVol!");
    goto Error;
  }
  for (int i = 0; i < numberOfFrames; ++i)
  {
    std::vector<unsigned char> vecVolSlice(nrrdData.begin() + (size_t)i * (size_t)srcRows * (size_t)srcColumns, nrrdData.begin() + ((size_t)i + 1) * (size_t)srcRows * (size_t)srcColumns);
    cudaStatus = hipMalloc(&deviceVecEachSlice[i], vecVolSlice.size() * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipMalloc failed for deviceVecEachSlice[%d]!", i);
      goto Error;
    }
    cudaStatus = hipMemcpy(deviceVecEachSlice[i], vecVolSlice.data(), vecVolSlice.size() * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipMemcpy failed at host to device for slice %d!", i);
      goto Error;
    }
  }
  cudaStatus = hipMemcpy(deviceVecVol, deviceVecEachSlice.data(), sizeof(unsigned char*) * numberOfFrames, hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed for deviceVecVol to device!");
    goto Error;
  }

  // Launch kernel
  dim3 threadsPerBlock(256);
  dim3 numBlocks((outputVecVol.size() + threadsPerBlock.x - 1) / threadsPerBlock.x);
  inverseScanConversionKernels <<< numBlocks, threadsPerBlock >>> (deviceOutVol, deviceVecVol, srcRows, srcColumns, numberOfFrames, dstRows, dstColumns, deviceIndexX, deviceIndexY);

  // Check for kernel launch errors
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "inverseScanConversionKernels launch failed: %s\n", hipGetErrorString(cudaStatus));
    goto Error;
  }

  // Synchronize kernel and check for errors
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching averageVectorNKernels!\n", cudaStatus);
    goto Error;
  }

  // Copy output vector from GPU buffer to host memory
  outputVecVol.resize((size_t)dstRows * (size_t)dstColumns * (size_t)numberOfFrames);
  cudaStatus = hipMemcpy(hostVecEachSlicePtr, deviceOutVol, sizeof(unsigned char*) * numberOfFrames, hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed for deviceVecVol to host!");
    goto Error;
  }
  for (int i = 0; i < numberOfFrames; ++i)
  {
    size_t sliceSize = dstRows * dstColumns;
    cudaStatus = hipMemcpy(outputVecVol.data() + i * sliceSize, hostVecEachSlicePtr[i], sliceSize * sizeof(unsigned char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipMemcpy failed at device to host for slice %d!", i);
      goto Error;
    }
  }


Error:
  hipFree(deviceIndexX);
  hipFree(deviceIndexY);
  for (int i = 0; i < numberOfFrames; ++i) {
    hipFree(deviceVecEachSlice[i]);
  }
  hipFree(deviceVecVol);
  hipFree(deviceOutVol);
  delete[] hostVecEachSlicePtr;
  return cudaStatus;
}

__global__ void bilinearInterpolationKernels
( unsigned char** deviceOutputVecVol,
  unsigned char** deviceVecVol,
  int srcRows,
  int srcCols,
  int numVectors,
  int dstRows,
  int dstCols,
  float* deviceIndexX,
  float* deviceIndexY,
  unsigned char* deviceMaskMeshVecPtr
)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= dstRows * dstCols)
    return;

  int row = (int)deviceIndexY[idx];
  int col = (int)deviceIndexX[idx];
  unsigned char mask = (deviceMaskMeshVecPtr != nullptr) ? deviceMaskMeshVecPtr[idx] : 1;

  if (row < 0 || col < 0 || row >= (srcRows - 1) || col >= (srcCols - 1) || mask == 0)
  {
    for (int vec = 0; vec < numVectors; ++vec)
    {
      deviceOutputVecVol[vec][idx] = 0;
    }
  }
  else
  {
    float row_frac = deviceIndexY[idx] - (float)row;
    float col_frac = deviceIndexX[idx] - (float)col;
    for (int vec = 0; vec < numVectors; ++vec)
    {
      float intensity_topleft = (float)deviceVecVol[vec][row * srcCols + col];
      float intensity_topright = (float)deviceVecVol[vec][row * srcCols + col + 1];
      float intensity_bottomleft = (float)deviceVecVol[vec][(row + 1) * srcCols + col];
      float intensity_bottomright = (float)deviceVecVol[vec][(row + 1) * srcCols + col + 1];
      // Bilinear interpolation
      deviceOutputVecVol[vec][idx] = (int)(((1.0f - row_frac) * ((1.0f - col_frac) * intensity_topleft + col_frac * intensity_topright)
        + row_frac * ((1.0f - col_frac) * intensity_bottomleft + col_frac * intensity_bottomright)) + 0.5f);
    }
  }
}

hipError_t bilinearInterpolationWithCuda
( std::vector<unsigned char>& outputVecVol,
  std::vector<unsigned char>& nrrdData,
  unsigned int srcRows,
  unsigned int srcColumns,
  unsigned int numberOfFrames,
  unsigned int dstRows,
  unsigned int dstColumns,
  std::vector<float> srcIndexWiseXVec,
  std::vector<float> srcIndexWiseYVec,
  std::vector<unsigned char> hostMaskMeshVec
)
{
  hipError_t cudaStatus;
  float* deviceIndexX;
  float* deviceIndexY;
  unsigned char* deviceMaskMeshVecPtr = nullptr;
  std::vector<unsigned char*> deviceVecEachSlice(numberOfFrames, nullptr);
  unsigned char** deviceVecVol;
  unsigned char** deviceOutVol;
  unsigned char** hostVecEachSlicePtr = new unsigned char* [numberOfFrames];

  // Select GPU
  cudaStatus = hipSetDevice(0);

  // malloc for deviceOutVol
  hipMalloc(&deviceOutVol, sizeof(unsigned char*) * numberOfFrames);
  for (int i = 0; i < numberOfFrames; ++i)
  {
    std::vector<unsigned char> zeroVecSlice((size_t)dstRows * (size_t)dstColumns, 0);
    hipMalloc(&deviceVecEachSlice[i], zeroVecSlice.size() * sizeof(unsigned char));
    hipMemcpy(deviceVecEachSlice[i], zeroVecSlice.data(), zeroVecSlice.size() * sizeof(unsigned char), hipMemcpyHostToDevice);
  }
  hipMemcpy(deviceOutVol, deviceVecEachSlice.data(), sizeof(unsigned char*) * numberOfFrames, hipMemcpyHostToDevice);
  // malloc and memcory for srcIndexWiseXVec and srcIndexWiseYVec
  hipMalloc(&deviceIndexX, srcIndexWiseXVec.size() * sizeof(float));
  hipMalloc(&deviceIndexY, srcIndexWiseYVec.size() * sizeof(float));

  if (!hostMaskMeshVec.empty())
  {
    hipMalloc(&deviceMaskMeshVecPtr, hostMaskMeshVec.size() * sizeof(unsigned char));
    hipMemcpy(deviceMaskMeshVecPtr, hostMaskMeshVec.data(), hostMaskMeshVec.size() * sizeof(unsigned char), hipMemcpyHostToDevice);
  }
  hipMemcpy(deviceIndexX, srcIndexWiseXVec.data(), srcIndexWiseXVec.size() * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceIndexY, srcIndexWiseYVec.data(), srcIndexWiseYVec.size() * sizeof(float), hipMemcpyHostToDevice);

  // malloc and memcory for nrrddata
  hipMalloc(&deviceVecVol, sizeof(unsigned char*) * numberOfFrames);
  for (int i = 0; i < numberOfFrames; ++i)
  {
    std::vector<unsigned char> vecVolSlice(nrrdData.begin() + (size_t)i * (size_t)srcRows * (size_t)srcColumns, nrrdData.begin() + ((size_t)i + 1) * (size_t)srcRows * (size_t)srcColumns);
    hipMalloc(&deviceVecEachSlice[i], vecVolSlice.size() * sizeof(unsigned char));
    hipMemcpy(deviceVecEachSlice[i], vecVolSlice.data(), vecVolSlice.size() * sizeof(unsigned char), hipMemcpyHostToDevice);
  }
  hipMemcpy(deviceVecVol, deviceVecEachSlice.data(), sizeof(unsigned char*) * numberOfFrames, hipMemcpyHostToDevice);

  // Launch kernel
  // std::vector<int> numTreadVec = arangeInt(32, 1024, 32); // pick 832 from 32 - 1024
  int numThreads = 832;
  //auto start_bilinintp = std::chrono::high_resolution_clock::now();
  dim3 threadsPerBlock(numThreads);
  dim3 numBlocks((outputVecVol.size() + threadsPerBlock.x - 1) / threadsPerBlock.x);
  bilinearInterpolationKernels << < numBlocks, threadsPerBlock >> > (deviceOutVol, deviceVecVol, srcRows, srcColumns, numberOfFrames, dstRows, dstColumns, deviceIndexX, deviceIndexY, deviceMaskMeshVecPtr);
  hipDeviceSynchronize();
  //auto end_bilinintp = std::chrono::high_resolution_clock::now();
  //std::chrono::duration<double, std::milli> duration_bilinintp = end_bilinintp - start_bilinintp;
  //std::cout << "Elapse time for the mask bilinear interpolation(in helper, only kernel), gpu: " << duration_bilinintp.count() << " msec " << " numThreads: "<< numThreads  <<"\n";

  // Copy output vector from GPU buffer to host memory
  outputVecVol.resize((size_t)dstRows * (size_t)dstColumns * (size_t)numberOfFrames);
  hipMemcpy(hostVecEachSlicePtr, deviceOutVol, sizeof(unsigned char*) * numberOfFrames, hipMemcpyDeviceToHost);
  for (int i = 0; i < numberOfFrames; ++i)
  {
    size_t sliceSize = dstRows * dstColumns;
    hipMemcpy(outputVecVol.data() + i * sliceSize, hostVecEachSlicePtr[i], sliceSize * sizeof(unsigned char), hipMemcpyDeviceToHost);
  }

  hipFree(deviceIndexX);
  hipFree(deviceIndexY);
  if (deviceMaskMeshVecPtr != nullptr)
  {
    hipFree(deviceMaskMeshVecPtr);
  }
  for (int i = 0; i < numberOfFrames; ++i)
  {
    hipFree(deviceVecEachSlice[i]);
  }
  hipFree(deviceVecVol);
  hipFree(deviceOutVol);
  delete[] hostVecEachSlicePtr;
  return cudaStatus;
}

