
#include "hip/hip_runtime.h"


#include <stdio.h>

void addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    addWithCuda(c, a, b, arraySize);
    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);
    hipDeviceReset();
    return 0;
}

void addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    
    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
    hipGetLastError();
    hipDeviceSynchronize();
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}
