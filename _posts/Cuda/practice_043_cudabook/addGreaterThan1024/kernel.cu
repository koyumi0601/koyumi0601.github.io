
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
    cudaStatus = hipGetLastError();
    cudaStatus = hipDeviceSynchronize();
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
